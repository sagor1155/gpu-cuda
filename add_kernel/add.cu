/*
 * How to compile (assume cuda is installed at /usr/local/cuda/)
 *   nvcc add.cu 
 *   ./a.out 
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>

__global__ void add_kernel(int* a, int* b, int*c){
	*c = *a + *b;
}

int main(void)
{
	printf("My First CUDA Application\n");
	int a, b, c;
	int *d_a, *d_b, *d_c;
	a = 10; b=20; c=0;
	int size = sizeof(int);

	// Allocate space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	// Copy inputs to device
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	float time;
	hipEvent_t start, stop;
	
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	
	// Launch add() kernel on GPU
	add_kernel<<<1, 1>>>(d_a, d_b, d_c);
	
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	
	printf("Time to generate:  %3.1f ms \n", time);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Copy result back to host
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	printf("Result is: %d\n", c);

	// Cleanup
	if(d_a) hipFree(d_a);
	if(d_b) hipFree(d_b);
	if(d_c) hipFree(d_c);

	return 0;
}
